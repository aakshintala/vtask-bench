#include "hip/hip_runtime.h"
#include <assert.h>
#include <cmath>
#include <cstdio>
#include <iomanip>
#include <iostream>
#include <iterator>
#include <sys/stat.h>
#include <sys/time.h>
#include <sys/types.h>
#include <sys/wait.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

using namespace std;

inline hipError_t cudaCheckError(hipError_t retval, const char *txt,
                                  const char *file, int line) {
#ifdef DEBUG
  std::cout << "[cuda] " << txt << std::endl;

  if (retval != hipSuccess) {
    std::cout << "[cuda] error" << retval << " " << hipGetErrorString(retval)
              << std::endl;
    std::cout << "[cuda] " << file << " " << line << std::endl;
  }
#endif

  return retval;
}

#define CUDACER(x) cudaCheckError((x), #x, __FILE__, __LINE__)
#define CUDA_ASSERT(x) assert(CUDACER(x) == hipSuccess)

// Wait until the application notifies us that it has completed queuing up the
// experiment, or timeout and exit, allowing the application to make progress
__global__ void delay(volatile int *flag, uint64_t cyclesToSpin = 10000000) {
  uint64_t startClock = clock64();
  while (*flag == 0 && clock64() - startClock <= cyclesToSpin) {
    continue;
  }
}

// This kernel just occupies the GPU for the specified number of cycles
__global__ void incrementBufferAndSpin(int *buffer, uint64_t num_elems,
                                       uint64_t cyclesToSpin = 10000000) {
  uint64_t startClock = clock64();

  while (clock64() - startClock <= cyclesToSpin) {
    continue;
  }
}

auto getGpuClockRate() -> int {
  hipDeviceProp_t prop;
  CUDA_ASSERT(hipGetDeviceProperties(&prop, 0));
  std::cout << "Device properties:\n"
            << "Name: " << prop.name << std::endl
            << "pciBusID:" << std::hex << prop.pciBusID << std::endl
            << "pciDeviceID:" << std::hex << prop.pciDeviceID << std::endl
            << "pciDomainID:" << std::hex << prop.pciDomainID << std::endl
            << std::dec << std::endl;
  return prop.clockRate;
}

void copyAndSpin(uint64_t numElems, size_t objectSize,
                 uint64_t computeTimeMicroseconds) {
  // We use the first GPU for our experiments
  hipSetDevice(0);

  uint64_t bufferSize = numElems * objectSize;

  // Allocate a pinned buffer on the host to copy from
  int *hostBuffer;
  CUDA_ASSERT(hipHostMalloc(&hostBuffer, bufferSize));
  for (int i = 0; i < bufferSize / sizeof(int); ++i)
    hostBuffer[i] = i;

  // Device side buffer to copy to.
  int *deviceBuffer;
  CUDA_ASSERT(hipMalloc(&deviceBuffer, bufferSize));

  // We use this flag as a sort of release gate for Async events queued on a
  // stream. Primarily used so we can control when we set start and stop events
  // to make time measurements. See the 'delay' kernel.
  volatile int *flag = NULL;
  CUDA_ASSERT(
      hipHostAlloc((void **)&flag, sizeof(*flag), hipHostMallocPortable));

  // Stream that we enqueue our Async operations to.
  // Using a stream theoretically enables us to avoid measuring CPU enqueue time
  hipStream_t stream;
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

  // Start and Stop events allow us to time the time it took to do the work
  hipEvent_t start, stop;
  CUDA_ASSERT(hipEventCreate(&start));
  CUDA_ASSERT(hipEventCreate(&stop));

#ifdef PROFILE
  pid_t pid = fork();
  if (pid == 0) {
    int childOutFD = open("./cpu-utilization", O_WRONLY | O_CREAT,
                          S_IRUSR | S_IWUSR | S_IRGRP | S_IROTH);
    dup2(childOutFD, STDOUT_FILENO);
    dup2(childOutFD, STDERR_FILENO);
    close(childOutFD);
    execl("./cpu-stat", "cpu-stat");
    exit(EXIT_SUCCESS);
  } else {
#endif // PROFILE
    CUDA_ASSERT(hipStreamSynchronize(stream));

    // Block the stream until all the work is queued up
    // DANGER! - hipMemcpyAsync may infinitely block waiting for
    // room to push the operation, so keep the number of repetitions
    // relatively low.  Higher repetitions will cause the delay kernel
    // to timeout and lead to unstable results.
    *flag = 0;
    delay<<<1, 1, 0, stream>>>(flag);

    // Figure out what the max numbers for blockSize and numBlocks can be on
    // this GPU.
    int blockSize = 128;
    int numBlocks = 1024;
    CUDA_ASSERT(hipOccupancyMaxPotentialBlockSize(&numBlocks, &blockSize,
                                                   incrementBufferAndSpin));

    uint64_t computeTimeCycles =
        (computeTimeMicroseconds / 1e6) * getGpuClockRate() * 1e3;
    std::cout << "Compute Time Cycles per element = " << computeTimeCycles
              << std::endl;

    // Enqueue GPU Commands to the stream; won't be executed until we set flag.
    CUDA_ASSERT(hipEventRecord(start, stream));
    for (int e = 0; e < numElems; e++) {
      uint64_t offset = e * objectSize / sizeof(int);
      CUDA_ASSERT(hipMemcpyAsync((void *)(hostBuffer + offset),
                                  (const void *)(deviceBuffer + offset),
                                  objectSize, hipMemcpyDeviceToHost, stream));
      // This kernel increments each element in the buffer and then spins until
      // the desired number of cycles have elapsed.
      incrementBufferAndSpin<<<numBlocks, blockSize, 0, stream>>>(
          (int *)deviceBuffer + offset, objectSize / sizeof(int),
          computeTimeCycles);

      CUDA_ASSERT(hipMemcpyAsync((void *)(deviceBuffer + offset),
                                  (const void *)(hostBuffer + offset),
                                  objectSize, hipMemcpyHostToDevice, stream));
    }
    CUDA_ASSERT(hipEventRecord(stop, stream));

    // Release the queued events and wait on stream synchronization event.
    *flag = 1;
    CUDA_ASSERT(hipStreamSynchronize(stream));

    // Measure elapsed time.
    float time_ms;
    hipEventElapsedTime(&time_ms, start, stop);

#ifdef PROFILE
    kill(pid, SIGTERM);
    int status;
    waitpid(pid, &status, 0);
#endif // PROFILE

    std::cout << "Transferred " << bufferSize / (double)1e9 << " GB of data in "
              << time_ms << " ms." << std::endl
              << "Number of chunks = " << numElems << "." << std::endl
              << "Chunk size = " << objectSize << " B" << std::endl
              << "Per chunk compute time =" << computeTimeMicroseconds << " us."
              << std::endl;

    // Free buffers and destroy stream & events
    CUDA_ASSERT(hipEventDestroy(stop));
    CUDA_ASSERT(hipEventDestroy(start));
    CUDA_ASSERT(hipStreamDestroy(stream));
    CUDA_ASSERT(hipFree(deviceBuffer));
    CUDA_ASSERT(hipHostFree(hostBuffer));
    CUDA_ASSERT(hipHostFree((void *)flag));

#ifdef PROFILE
  }
#endif // PROFILE
}

void panicIfNoGPU() {
  std::cout << "Making sure there is at least one GPU on this system. Will use "
               "GPU 0 if there are multiple."
            << std::endl;
  int numGPUs = 0;
  CUDA_ASSERT(hipGetDeviceCount(&numGPUs));
  assert(numGPUs != 0);
}

int main(int argc, char **argv) {
  uint64_t queueDepth = 500000; // 1 million
  size_t objectSize = 1024 * sizeof(int);
  int computeTimeMicroseconds = 10;

  // process command line args
  for (int i = 1; i < argc; i++) {
    if (0 == strcmp(argv[i], "-h")) {
      std::cerr << "Usage:" << argv[0] << " [OPTION]..." << std::endl
                << "Options:" << std::endl
                << "\t-h\tDisplay this Help menu" << std::endl
                << "\t-q\tNumber of Chunks ()" << std::endl
                << "\t-s\tChunk size (in increments of sizeof(int))"
                << std::endl
                << "\t-t\tPer chunk compute time to simulate (us)" << std::endl
                << std::endl
                << std::endl;
      return EXIT_FAILURE;
    } else if (0 == strcmp(argv[i], "-q")) {
      queueDepth = atoi(argv[i + 1]);
    } else if (0 == strcmp(argv[i], "-s")) {
      objectSize = atoi(argv[i + 1]) * sizeof(int);
    } else if (0 == strcmp(argv[i], "-t")) {
      computeTimeMicroseconds = atoi(argv[i + 1]);
    }
  }

  panicIfNoGPU();

  std::cout << "Synthetic GPU data movement and compute benchmark.\n";
  std::cout << "Moving " << queueDepth * objectSize / 1000 / 1000
            << " MB of data." << std::endl;
  std::cout << "Simulated compute time: " << computeTimeMicroseconds << " us."
            << std::endl;

  copyAndSpin(queueDepth, objectSize, computeTimeMicroseconds);

  exit(EXIT_SUCCESS);
}
